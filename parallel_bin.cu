#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>
#include <vector>

#include <hipcub/hipcub.hpp>

#include "Util_cuda.cuh"

#include "parallel_bin.cuh"


void ParallelBin::makeBin(int totalBinEventCount, const int Events3D, BinParameters *bin, SetupParameters *setup, ReductionVars *reduce)
{
    //Setup device specific values
    
    //Current CUDA device ID
    int devId;
    hipGetDevice(&devId);

    //Size of the thread blocks
    int blockSize = 256;
    //Number of device multiprocessing units available
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);
    //Effective size of the input reduction variables, depending on the number of events to simulate
    const int reduceSize = std::min({warpSize * numSMs * blockSize, totalBinEventCount});

    
    //State for the random generators 
    hiprandState *state;
    hipMalloc(&state, sizeof(hiprandState));
    // distributions -> https://docs.nvidia.com/cuda/hiprand/device-api-overview.html#distributions
    //Setup the initial state based on the system clock
    
    std::chrono::time_point<std::chrono::system_clock> now = std::chrono::system_clock::now();
    auto duration = now.time_since_epoch();

    unsigned int seed = std::chrono::duration_cast<std::chrono::nanoseconds>(duration).count();

    Util_CUDA::setupRand<<<1,1>>>(state, seed);
    
    
    //Initialize device side pointers for the simulation variables

    //Bin variables
    BinParameters *binCUDA;
    //Setup variables
    SetupParameters *setupCUDA;
    //Summation reduction variables
    ReductionVars *reduceCUDA;


    //Allocate structures in the device
    hipMalloc(&binCUDA, sizeof(BinParameters));
    hipMalloc(&setupCUDA, sizeof(SetupParameters));
    hipMalloc(&reduceCUDA, reduceSize * sizeof(ReductionVars));

    //Copy the initial values from the host
    hipMemcpy(binCUDA, bin, sizeof(BinParameters), hipMemcpyHostToDevice);
    hipMemcpy(setupCUDA, setup, sizeof(SetupParameters), hipMemcpyHostToDevice);
    
    for(int i = 0; i < reduceSize; i++)
    {
        hipMemcpy(&reduceCUDA[i], reduce, sizeof(ReductionVars), hipMemcpyHostToDevice);
    }

    
    //Execute the bin simulation
    makeBinKernel<<<warpSize * numSMs, blockSize>>>(totalBinEventCount, state, binCUDA, setupCUDA, reduceCUDA);

    //Synchronize the device to perform reduction
    hipDeviceSynchronize();
    

    // Perform the reduction using CUB
    
    //Pointer where to store the reduce sum
    ReductionVars *d_sum;
    //allocate it
    hipMalloc(&d_sum, sizeof(ReductionVars));
    
    
    // Determine temporary device storage requirements
    
    //Pointer to the temporary reduction storage
    void *d_temp_storage = NULL;
    //Size of the temporary reduction storage
    size_t temp_storage_bytes = 0;

    //Initialize the size of the temporary storage required
    // hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, reduceCUDA, d_sum, reduceSize);
    //Allocate the temporary storage
    // hipMalloc(&d_temp_storage, temp_storage_bytes);
    //Perform the sum reduction
    // hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, reduceCUDA, d_sum, reduceSize);

    
    hipMemcpy(bin, binCUDA, sizeof(BinParameters), hipMemcpyDeviceToHost);
    hipMemcpy(reduce, d_sum, sizeof(ReductionVars), hipMemcpyDeviceToHost);

    // Free memory on the GPU
    hipFree(binCUDA);
    hipFree(reduceCUDA);
    hipFree(setupCUDA);
    
    hipFree(d_temp_storage);
    hipFree(d_sum);
}


__global__ static void makeBinKernel(int totalBinEventCount, hiprandState *randState, 
                            ParallelBin::BinParameters *bin, ParallelBin::SetupParameters *setup, ParallelBin::ReductionVars *reduce)
{
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    int stride = blockDim.x * gridDim.x;


    double teta_table_thread = bin->teta_table_thread; double p = bin->p; double tetadir = bin->tetadir; double sin_tetadir = bin->sin_tetadir;
    double cos_tetadir = bin->cos_tetadir; double tan_tetadir = bin->tan_tetadir; double cos_tetadirCry1 = bin->cos_tetadirCry1;
    double fidir = bin->fidir; double sin_fidir = bin->sin_fidir; double cos_fidir = bin->cos_fidir; double tan_fidir = bin->tan_fidir;
    double cos_fidirtilt = bin->cos_fidirtilt; double z = bin->z; double y = bin->y; double r = bin->r; double tetap = bin->tetap;
    double sin_tetap = bin->sin_tetap; double cos_tetap = bin->cos_tetap; double var_temp = bin->var_temp; double y_pro_C1 = bin->y_pro_C1;
    double z_pro_C1 = bin->z_pro_C1; double rx = bin->rx; double ry = bin->ry; double rz = bin->rz; double Costeta_CHC = bin->Costeta_CHC;
    double Sinteta_CHC = bin->Sinteta_CHC; double n1x = bin->n1x; double n1y = bin->n1y; double angle = bin->angle; double r2x = bin->r2x; double r2y = bin->r2y;
    double r2z = bin->r2z; double lamda = bin->lamda; double tetabra1 = bin->tetabra1; bool first_crystal_reach = bin->first_crystal_reach; double rx_rot = bin->rx_rot;
    double ry_rot = bin->ry_rot; double cos_fidirtilt2_para = bin->cos_fidirtilt2_para; double cos_fidirtilt2_anti = bin->cos_fidirtilt2_anti; double corr_dis = bin->corr_dis;
    double sin_tetatab_del_dir = bin->sin_tetatab_del_dir; double tetabra2 = bin->tetabra2; double n2x_para = bin->n2x_para; double n2y_para = bin->n2y_para;
    double r3x = bin->r3x; double r3y = bin->r3y; double r3z = bin->r3z; bool sec_crystal_Parallel_reach = bin->sec_crystal_Parallel_reach; double rx_rot_sec = bin->rx_rot_sec;
    double ry_rot_sec = bin->ry_rot_sec; double tetadir_det = bin->tetadir_det; double tan_tetadir_det = bin->tan_tetadir_det; double fidir_det = bin->fidir_det;
    double tan_fidir_det = bin->tan_fidir_det; double corr_dis_d_pa = bin->corr_dis_d_pa; double y_pro_C1_d_pa = bin->y_pro_C1_d_pa; double cos_tetap_det = bin->cos_tetap_det;
    double sin_tetap_det = bin->sin_tetap_det; double r_det = bin->r_det; double y_det = bin->y_det; double z_det = bin->z_det; double n2x_anti = bin->n2x_anti;
    double n2y_anti = bin->n2y_anti; bool sec_crystal_Antiparallel_reach = bin->sec_crystal_Antiparallel_reach; int total_current_bins = bin->total_current_bins;
    double cos_tetadir_det = bin->cos_tetadir_det; double cos_fidir_det = bin->cos_fidir_det;

    double table_noise_width = (setup->table_resolution / 2.355) / 6.0;

    for (int i = idx; i < totalBinEventCount; i += stride)
    {
        //Temporary event to show in the 3D view
        //If we have less than maxEventNum we just append otherwise we see
        //this temporary event stored reaches the exit before appending.
        //Each triple of values is 1 point and each event will have 4 points.
        //Source - Crystal1 - Crystal2 - Detector
        double *tmpEvent_para = new double[12];
        double *tmpEvent_anti = new double[12];
        //current size of the temporary event (0, 3, 6, 9, 12)
        int tmpSize_para = 0;
        int tmpSize_anti = 0;

        if(setup->Make_Horizontal){
            p = setup->del_teta_L * hiprand_uniform_double(randState + idx + stride) + setup->teta_min_L;
            
            tetadir = p;
        }else
            tetadir = 0;


        sin_tetadir = sin(tetadir);
        cos_tetadir = cos(tetadir);
        tan_tetadir = sin_tetadir / cos_tetadir;

        if(setup->make_table_noise && setup->table_resolution >= 0.01)
        {
            teta_table_thread = setup->teta_table;
            teta_table_thread += table_noise_width * hiprand_normal_double(randState + idx + stride);
        }

        cos_tetadirCry1 = cos(tetadir + teta_table_thread * convrad);


        if(setup->Make_Vertical){
            p = setup->del_fi_L * hiprand_uniform_double(randState + idx + stride) + setup->fi_min_L;
            fidir = p;
        }else
            fidir = setup->xsi * convrad;


        sin_fidir = sin(fidir);
        cos_fidir = cos(fidir);
        tan_fidir = sin_fidir / cos_fidir;
        cos_fidirtilt = cos(fidir + setup->tilt_C1_temp);


        if(setup->type_source[0] == 'P'){
            z = setup->S_shi_ver_B;
            y = setup->S_shi_hor_B;
            r = sqrt(pow(z, 2) + pow(y, 2));

            if(setup->S_shi_ver_B == 0 && setup->S_shi_hor_B == 0){
                sin_tetap = 0;
                cos_tetap = 1;
            }else if(setup->S_shi_hor_B == 0){
                sin_tetap = 1;
                cos_tetap = 0;
            }else{
                var_temp = z / y;
                if(y > 0){
                    sin_tetap = var_temp / sqrt(1 + pow(var_temp, 2));
                    cos_tetap = 1 / sqrt(1 + pow(var_temp, 2));
                }else{
                    sin_tetap = - var_temp / sqrt(1 + pow(var_temp, 2));
                    cos_tetap = - 1 / sqrt(1 + pow(var_temp, 2));
                }
            }

        }else if(setup->type_source[0] == 'U' && setup->type_source[1] == 'C'){
            r = setup->S_sour_2 + 1;
            while(r > setup->S_sour_2){
                z = hiprand_uniform_double(randState + idx + stride) * setup->S_sour - setup->S_sour_2;
                y = hiprand_uniform_double(randState + idx + stride) * setup->S_sour - setup->S_sour_2;
                r = sqrt(pow(z, 2) + pow(y, 2));
            }


            z += setup->S_shi_ver_B;
            y += setup->S_shi_hor_B;

            if(y != 0)
                var_temp = z / y;
            else
                var_temp = 0;

            r = sqrt(pow(z, 2) + pow(y, 2));

            if(y > 0){
                sin_tetap = var_temp / sqrt(1 + pow(var_temp, 2));
                cos_tetap = 1 / sqrt(1 + pow(var_temp, 2));
            }else{
                sin_tetap = - var_temp / sqrt(1 + pow(var_temp, 2));
                cos_tetap = - 1 / sqrt(1 + pow(var_temp, 2));
            }
        }else if(setup->type_source[0] == 'G'){
            p = 2 * CR_CUDART_PI * hiprand_uniform_double(randState + idx + stride);
            tetap = p;
            sin_tetap = sin(tetap);
            cos_tetap = cos(tetap);
            r = setup->S_aper_R_2 + 1;

            while(r > setup->S_aper_R_2)
                r = Util_CUDA::GaussianBox(randState + idx + stride, setup->S_aper_var_2, 0);

            if(! (setup->S_shi_ver_B == 0 && setup->S_shi_hor_B == 0)){
                z = r * sin_tetap + setup->S_shi_ver_B;
                y = r * cos_tetap + setup->S_shi_hor_B;
                var_temp = z / y;
                r = sqrt(pow(z, 2) + pow(y, 2));

                if(y > 0){
                    sin_tetap = var_temp / sqrt(1 + pow(var_temp, 2));
                    cos_tetap = 1 / sqrt(1 + pow(var_temp, 2));
                }else{
                    sin_tetap = - var_temp / sqrt(1 + pow(var_temp, 2));
                    cos_tetap = - 1 / sqrt(1 + pow(var_temp, 2));
                }
            }

        }else if(setup->type_source[0] == 'U' && setup->type_source[0] == 'R'){
            z = hiprand_uniform_double(randState + idx + stride) * setup->z_sour - setup->z_sour_2;
            y = hiprand_uniform_double(randState + idx + stride) * setup->y_sour - setup->y_sour_2;

            z += setup->S_shi_ver_B;
            y += setup->S_shi_hor_B;
            var_temp = z / y;
            r = sqrt(pow(z, 2) + pow(y, 2));

            if(y > 0){
                sin_tetap = var_temp / sqrt(1 + pow(var_temp, 2));
                cos_tetap = 1 / sqrt(1 + pow(var_temp, 2));
            }else{
                sin_tetap = - var_temp / sqrt(1 + pow(var_temp, 2));
                cos_tetap = - 1 / sqrt(1 + pow(var_temp, 2));
            }
        }

        double2 yz = Util_CUDA::getYZ(r, sin_tetap, cos_tetap, tan_tetadir, tan_fidir, setup->LT_aper_Db);

        y = yz.x;
        z = yz.y;

        var_temp = pow(y - setup->S_shi_hor_A, 2) + pow(z - setup->S_shi_ver_A, 2);

        if(var_temp < setup->S_aper_sqr){

            if(setup->make_image_plates){
                Util_CUDA::MakeParameters *pars = new Util_CUDA::MakeParameters(
                                    reduce[idx].max_plot_x,
                                    reduce[idx].max_plot_y,
                                    reduce[idx].hist_image_plate_source,
                                    reduce[idx].hist_image_plate_crystal1,
                                    reduce[idx].hist_image_plate_crystal2_para,
                                    reduce[idx].hist_image_plate_crystal2_anti,
                                    reduce[idx].hist_image_plate_detc_para,
                                    reduce[idx].hist_image_plate_detc_anti,
                                    reduce[idx].max_hist,
                                    reduce[idx].counts_sour,
                                    reduce[idx].counts_C1,
                                    reduce[idx].counts_C2_para,
                                    reduce[idx].counts_detc_para,
                                    reduce[idx].counts_C2_anti,
                                    reduce[idx].counts_detc_anti
                );

                Util_CUDA::Make(1, y, z, pars);

                //Event point at source
                tmpEvent_para[0] = 0; //X
                tmpEvent_para[1] = y; //Y
                tmpEvent_para[2] = z; //Z
                tmpSize_para = 3;

                tmpEvent_anti[0] = 0; //X
                tmpEvent_anti[1] = y; //Y
                tmpEvent_anti[2] = z; //Z
                tmpSize_anti = 3;
            }


            r = sqrt(pow(y, 2) + pow(z, 2));

            double2 yz = Util_CUDA::getYZ(r, sin_tetap, cos_tetap, tan_tetadir, tan_fidir, setup->LT_aper_Db);

            y = yz.x;
            z = yz.y;


            y_pro_C1 = y * cos_tetadir / cos_tetadirCry1;
            z_pro_C1 = z * cos_fidir / cos_fidirtilt;

            if(y_pro_C1 < setup->y_max_C1 && y_pro_C1 > setup->y_min_C1 && z_pro_C1 < setup->z_max_C1 && z_pro_C1 > setup->z_min_C1){
                if(!setup->make_imageC1_After_refle){
                    if(setup->make_image_plates){
                        Util_CUDA::MakeParameters *pars = new Util_CUDA::MakeParameters(
                                    reduce[idx].max_plot_x,
                                    reduce[idx].max_plot_y,
                                    reduce[idx].hist_image_plate_source,
                                    reduce[idx].hist_image_plate_crystal1,
                                    reduce[idx].hist_image_plate_crystal2_para,
                                    reduce[idx].hist_image_plate_crystal2_anti,
                                    reduce[idx].hist_image_plate_detc_para,
                                    reduce[idx].hist_image_plate_detc_anti,
                                    reduce[idx].max_hist,
                                    reduce[idx].counts_sour,
                                    reduce[idx].counts_C1,
                                    reduce[idx].counts_C2_para,
                                    reduce[idx].counts_detc_para,
                                    reduce[idx].counts_C2_anti,
                                    reduce[idx].counts_detc_anti
                        );
                        
                        Util_CUDA::Make(2, y_pro_C1, z_pro_C1, pars);
                        
                        //Event point at the first crystal
                        tmpEvent_para[3] = 0; //X
                        tmpEvent_para[4] = y_pro_C1; //Y
                        tmpEvent_para[5] = z_pro_C1; //Z
                        tmpSize_para = 6;

                        tmpEvent_anti[3] = 0; //X
                        tmpEvent_anti[4] = y_pro_C1; //Y
                        tmpEvent_anti[5] = z_pro_C1; //Z
                        tmpSize_anti = 6;
                    }
                }

                rx = cos_fidir * cos_tetadir;
                ry = cos_fidir * sin_tetadir;
                rz = sin_fidir;


                if(setup->Curve_crystall){
                    double2 corrRes = Util_CUDA::horCorr(
                        y_pro_C1,
                        setup->y_max_C1,
                        z_pro_C1,
                        setup->z_max_C1,
                        true,
                        setup->R_cur_crys_1,
                        setup->R_cur_crys_2);

                    Costeta_CHC = corrRes.x;
                    Sinteta_CHC = corrRes.y;

                    n1x = setup->n1x_temp * Costeta_CHC + setup->n1y_temp * Sinteta_CHC;
                    n1y = setup->n1y_temp * Costeta_CHC - setup->n1x_temp * Sinteta_CHC;
                }

                double4 angleRes = Util_CUDA::getFullAngle(rx, ry, rz, n1x, n1y, setup->n1z);
                angle = angleRes.x;
                r2x = angleRes.y;
                r2y = angleRes.z;
                r2z = angleRes.w;

                Util_CUDA::EnergyParameters pars = {
                    setup->make_more_lines,
                    setup->Do_background,
                    setup->p1_ener,
                    setup->p2_ener,
                    setup->p3_ener,
                    setup->Energy_spectrum_vectors,
                    setup->picks[5],
                    setup->gauss_Doop
                };

                lamda = Util_CUDA::getEnergy(randState + idx + stride, setup->a_lamds_uni, setup->b_lamds_uni, setup->tw_d1_para, pars);
                
                tetabra1 = asin(lamda / setup->tw_d1_para);

                bool poliP;

                if(setup->mka_poli)
                {
                    if(hiprand_uniform_double(randState + idx + stride) < setup->relationP_S)
                        poliP = true;
                    else
                        poliP = false;
                }
                else
                {
                    poliP = false;
                }

                Util_CUDA::ReflectionParameters parsR = {
                    setup->available_energies,
                    setup->min_angle_resp,
                    setup->max_angle_resp,
                    setup->mka_poli,
                    setup->Crystal_Responces
                };

                first_crystal_reach = Util_CUDA::getReflection(randState + idx + stride,
                        angle,
                        tetabra1,
                        lamda,
                        false,
                        parsR,
                        poliP);

                if(first_crystal_reach){
                    if(setup->make_imageC1_After_refle){
                        if(setup->make_image_plates){
                            Util_CUDA::MakeParameters *pars = new Util_CUDA::MakeParameters(
                                    reduce[idx].max_plot_x,
                                    reduce[idx].max_plot_y,
                                    reduce[idx].hist_image_plate_source,
                                    reduce[idx].hist_image_plate_crystal1,
                                    reduce[idx].hist_image_plate_crystal2_para,
                                    reduce[idx].hist_image_plate_crystal2_anti,
                                    reduce[idx].hist_image_plate_detc_para,
                                    reduce[idx].hist_image_plate_detc_anti,
                                    reduce[idx].max_hist,
                                    reduce[idx].counts_sour,
                                    reduce[idx].counts_C1,
                                    reduce[idx].counts_C2_para,
                                    reduce[idx].counts_detc_para,
                                    reduce[idx].counts_C2_anti,
                                    reduce[idx].counts_detc_anti
                            );
                            
                            Util_CUDA::Make(2, y_pro_C1, z_pro_C1, pars);
                        }
                    }
                    
                    rx_rot = setup->cos_tetartab * r2x + setup->sin_tetartab * r2y;
                    ry_rot = -setup->sin_tetartab * r2x + setup->cos_tetartab * r2y;

                    var_temp = ry_rot / rx_rot;

                    tetadir = atan(var_temp);
                    tan_tetadir = var_temp;
                    cos_tetadir = cos(tetadir);


                    var_temp = r2z / sqrt(pow(rx_rot, 2) + pow(ry_rot, 2));
                    fidir = atan(var_temp);
                    tan_fidir = var_temp;
                    cos_fidir = cos(fidir);
                    cos_fidirtilt2_para = cos(fidir + setup->tilt_C2_para_temp);
                    cos_fidirtilt2_anti = cos(fidir + setup->tilt_C2_anti_temp);

                    corr_dis = y_pro_C1 * setup->cos_difteC1_Ta;

                    y_pro_C1 = -y_pro_C1 * setup->sin_difteC1_Ta;


                    if(y_pro_C1 == 0){
                        cos_tetap = 0;
                        if(z_pro_C1 > 0)
                            sin_tetap = 1;
                        else
                            sin_tetap = -1;
                    }else{
                        var_temp = z_pro_C1 / y_pro_C1;
                        r = sqrt(pow(z_pro_C1, 2) + pow(y_pro_C1, 2));

                        if(y_pro_C1 > 0){
                            sin_tetap = var_temp / sqrt(1 + pow(var_temp, 2));
                            cos_tetap = 1 / sqrt(1 + pow(var_temp, 2));
                        }else{
                            sin_tetap = - var_temp / sqrt(1 + pow(var_temp, 2));
                            cos_tetap = - 1 / sqrt(1 + pow(var_temp, 2));
                        }

                    }

                    double2 yz = Util_CUDA::getYZ(r, sin_tetap, cos_tetap, tan_tetadir, tan_fidir, setup->dist_Cr1_Cr2_Db - corr_dis);

                    y = yz.x;
                    z = yz.y;


                    if(setup->see_para){
                        sin_tetatab_del_dir = sin(setup->tetaref - setup->delrot + tetadir);
                        y_pro_C1 = y * cos_tetadir / sin_tetatab_del_dir;
                        z_pro_C1 = z * cos_fidir / cos_fidirtilt2_para;
                        
                        if(y_pro_C1 < setup->y_max_C2 && y_pro_C1 > setup->y_min_C2 && z_pro_C1 < setup->z_max_C2 && z_pro_C1 > setup->z_min_C2){

                            if(setup->make_image_plates){
                                if(!setup->make_imageC2_After_refle){
                                    Util_CUDA::MakeParameters *pars = new Util_CUDA::MakeParameters(
                                                reduce[idx].max_plot_x,
                                                reduce[idx].max_plot_y,
                                                reduce[idx].hist_image_plate_source,
                                                reduce[idx].hist_image_plate_crystal1,
                                                reduce[idx].hist_image_plate_crystal2_para,
                                                reduce[idx].hist_image_plate_crystal2_anti,
                                                reduce[idx].hist_image_plate_detc_para,
                                                reduce[idx].hist_image_plate_detc_anti,
                                                reduce[idx].max_hist,
                                                reduce[idx].counts_sour,
                                                reduce[idx].counts_C1,
                                                reduce[idx].counts_C2_para,
                                                reduce[idx].counts_detc_para,
                                                reduce[idx].counts_C2_anti,
                                                reduce[idx].counts_detc_anti
                                    );
                                    
                                    Util_CUDA::Make(3, y_pro_C1, z_pro_C1, pars);

                                    //Event point at second crystal in parallel
                                    tmpEvent_para[6] = 0; //X
                                    tmpEvent_para[7] = y_pro_C1; //Y
                                    tmpEvent_para[8] = z_pro_C1; //Z
                                    tmpSize_para = 9;
                                }
                            }

                            
                            tetabra2 = asin(lamda / setup->tw_d2_para);


                            if(setup->Curve_crystall){
                                double2 corrRes = Util_CUDA::horCorr(
                                    y_pro_C1,
                                    setup->y_max_C2,
                                    z_pro_C1,
                                    setup->z_max_C2,
                                    false,
                                    setup->R_cur_crys_1,
                                    setup->R_cur_crys_2);

                                Costeta_CHC = corrRes.x;
                                Sinteta_CHC = corrRes.y;

                                n2x_para = setup->n2x_para_temp * Costeta_CHC + setup->n2y_para_temp * Sinteta_CHC;
                                n2y_para = setup->n2y_para_temp * Costeta_CHC - setup->n2x_para_temp * Sinteta_CHC;
                            }


                            double4 angleRes = Util_CUDA::getFullAngle2(r2x, r2y, r2z, n2x_para, n2y_para, setup->n2z_para);
                            angle = angleRes.x;
                            r3x = angleRes.y;
                            r3y = angleRes.z;
                            r3z = angleRes.w;

                            
                            Util_CUDA::ReflectionParameters parsR = {
                                setup->available_energies,
                                setup->min_angle_resp,
                                setup->max_angle_resp,
                                setup->mka_poli,
                                setup->Crystal_Responces
                            };

                            sec_crystal_Parallel_reach = Util_CUDA::getReflection(randState + idx + stride,
                                    angle,
                                    tetabra2,
                                    lamda,
                                    false,
                                    parsR,
                                    poliP);
                            
                            if(sec_crystal_Parallel_reach){

                                if(setup->make_image_plates){
                                    if(setup->make_imageC2_After_refle){
                                        Util_CUDA::MakeParameters *pars = new Util_CUDA::MakeParameters(
                                                reduce[idx].max_plot_x,
                                                reduce[idx].max_plot_y,
                                                reduce[idx].hist_image_plate_source,
                                                reduce[idx].hist_image_plate_crystal1,
                                                reduce[idx].hist_image_plate_crystal2_para,
                                                reduce[idx].hist_image_plate_crystal2_anti,
                                                reduce[idx].hist_image_plate_detc_para,
                                                reduce[idx].hist_image_plate_detc_anti,
                                                reduce[idx].max_hist,
                                                reduce[idx].counts_sour,
                                                reduce[idx].counts_C1,
                                                reduce[idx].counts_C2_para,
                                                reduce[idx].counts_detc_para,
                                                reduce[idx].counts_C2_anti,
                                                reduce[idx].counts_detc_anti
                                        );
                                        
                                        Util_CUDA::Make(3, y_pro_C1, z_pro_C1, pars);
                                    }
                                }


                                rx_rot_sec = setup->cos_tetartabdete_para * r3x + setup->sin_tetartabdete_para * r3y;
                                ry_rot_sec = -setup->sin_tetartabdete_para * r3x + setup->cos_tetartabdete_para * r3y;

                                var_temp = ry_rot_sec / rx_rot_sec;



                                tetadir_det = atan(var_temp);
                                tan_tetadir_det = var_temp;
                                cos_tetadir_det = cos(tetadir_det);


                                var_temp = rz / sqrt(pow(rx_rot, 2) + pow(ry_rot, 2));
                                fidir_det = atan(var_temp);
                                tan_fidir_det = var_temp;
                                cos_fidir_det = cos(fidir_det);
                                cos_fidirtilt2_para = cos(fidir + setup->tilt_C2_para_temp);

                                corr_dis_d_pa = -y_pro_C1 * setup->cos_difteC2_det_para;
                                y_pro_C1_d_pa = -y_pro_C1 * setup->sin_difteC2_det_para;


                                if(y_pro_C1_d_pa == 0){
                                    cos_tetap_det = 0;
                                    if(z_pro_C1 > 0)
                                        sin_tetap_det = 1;
                                    else
                                        sin_tetap_det = -1;
                                }else{
                                    var_temp = z_pro_C1 / y_pro_C1_d_pa;
                                    r_det = sqrt(pow(z_pro_C1, 2) + pow(y_pro_C1_d_pa, 2));

                                    if(y_pro_C1_d_pa > 0){
                                        sin_tetap_det = var_temp / sqrt(1 + pow(var_temp, 2));
                                        cos_tetap_det = 1 / sqrt(1 + pow(var_temp, 2));
                                    }else{
                                        sin_tetap_det = - var_temp / sqrt(1 + pow(var_temp, 2));
                                        cos_tetap_det = - 1 / sqrt(1 + pow(var_temp, 2));
                                    }
                                }

                                double2 yz = Util_CUDA::getYZ(r_det, sin_tetap_det, cos_tetap_det, tan_tetadir_det, tan_fidir_det, setup->dist_Cr2_det_Db - corr_dis_d_pa);

                                y_det = yz.x;
                                z_det = yz.y;

                                if(y_det < setup->ydetc_2_max && y_det > setup->ydetc_2_min && z_det < setup->zdetc_2_max && z_det > setup->zdetc_2_min){

                                    if(setup->make_image_plates){
                                        Util_CUDA::MakeParameters *pars = new Util_CUDA::MakeParameters(
                                                reduce[idx].max_plot_x,
                                                reduce[idx].max_plot_y,
                                                reduce[idx].hist_image_plate_source,
                                                reduce[idx].hist_image_plate_crystal1,
                                                reduce[idx].hist_image_plate_crystal2_para,
                                                reduce[idx].hist_image_plate_crystal2_anti,
                                                reduce[idx].hist_image_plate_detc_para,
                                                reduce[idx].hist_image_plate_detc_anti,
                                                reduce[idx].max_hist,
                                                reduce[idx].counts_sour,
                                                reduce[idx].counts_C1,
                                                reduce[idx].counts_C2_para,
                                                reduce[idx].counts_detc_para,
                                                reduce[idx].counts_C2_anti,
                                                reduce[idx].counts_detc_anti
                                        );
                                        
                                        Util_CUDA::Make(4, y_det, z_det, pars);
                                        
                                        //Event point at detector in parallel
                                        tmpEvent_para[9] = 0; //X
                                        tmpEvent_para[10] = y_det; //Y
                                        tmpEvent_para[11] = z_det; //Z
                                        tmpSize_para = 12;

                                        if (bin->curr3DEventCount_para <= setup->number_events) {
                                            reduce[idx].eventsToTrace_paraCUDA[bin->curr3DEventCount_para] = tmpEvent_para;
                                            atomicAdd(&(bin->curr3DEventCount_para), 1);
                                        }
                                        else {
                                            for (int i = 0; i < bin->curr3DEventCount_para; ++i)
                                                reduce[idx].eventsToTrace_paraCUDA[i] = reduce[idx].eventsToTrace_paraCUDA[i + 1];
                                            
                                            reduce[idx].eventsToTrace_paraCUDA[bin->curr3DEventCount_para] = tmpEvent_para;
                                        }
                                    }

                                    reduce[idx].toint_para++;

                                    reduce[idx].energy_sum_para_thread += Convert_Ag_minusone_eV / lamda;
                                }
                                else {
                                    if (setup->make_image_plates) {
                                        //If the event does not reach the detector then only add when we have less than maxEventNum
                                        if (bin->curr3DEventCount_para <= setup->number_events && tmpSize_para >= 6) {
                                            reduce[idx].eventsToTrace_paraCUDA[bin->curr3DEventCount_para] = tmpEvent_para;
                                            atomicAdd(&(bin->curr3DEventCount_para), 1);
                                        }
                                    }
                                }
                            }
                            else {
                                if (setup->make_image_plates) {
                                    //If the event does not reach the detector then only add when we have less than maxEventNum
                                    if (bin->curr3DEventCount_para < setup->number_events && tmpSize_para >= 6) {
                                        reduce[idx].eventsToTrace_paraCUDA[bin->curr3DEventCount_para] = tmpEvent_para;
                                        atomicAdd(&(bin->curr3DEventCount_para), 1);
                                    }
                                }
                            }
                        }
                        else {
                            if (setup->make_image_plates) {
                                //If the event does not reach the detector then only add when we have less than maxEventNum
                                if (bin->curr3DEventCount_para < setup->number_events && tmpSize_para >= 6) {
                                    reduce[idx].eventsToTrace_paraCUDA[bin->curr3DEventCount_para] = tmpEvent_para;
                                    atomicAdd(&(bin->curr3DEventCount_para), 1);
                                }
                            }
                        }

                    }


                    if(setup->see_anti){


                        sin_tetatab_del_dir = sin(setup->tetaref + setup->delrot - tetadir);

                        y_pro_C1 = y * cos_tetadir / sin_tetatab_del_dir;
                        z_pro_C1 = z * cos_fidir / cos_fidirtilt2_anti;

                        
                        if(y_pro_C1 < setup->y_max_C2 && y_pro_C1 > setup->y_min_C2 && z_pro_C1 < setup->z_max_C2 && z_pro_C1 > setup->z_min_C2){


                            if(setup->make_imageC2_After_refle){
                                if(setup->make_image_plates){
                                    Util_CUDA::MakeParameters *pars = new Util_CUDA::MakeParameters(
                                                reduce[idx].max_plot_x,
                                                reduce[idx].max_plot_y,
                                                reduce[idx].hist_image_plate_source,
                                                reduce[idx].hist_image_plate_crystal1,
                                                reduce[idx].hist_image_plate_crystal2_para,
                                                reduce[idx].hist_image_plate_crystal2_anti,
                                                reduce[idx].hist_image_plate_detc_para,
                                                reduce[idx].hist_image_plate_detc_anti,
                                                reduce[idx].max_hist,
                                                reduce[idx].counts_sour,
                                                reduce[idx].counts_C1,
                                                reduce[idx].counts_C2_para,
                                                reduce[idx].counts_detc_para,
                                                reduce[idx].counts_C2_anti,
                                                reduce[idx].counts_detc_anti
                                    );
                                    
                                    Util_CUDA::Make(5, y_pro_C1, z_pro_C1, pars);

                                    //Event point at second crystal in antiparallel
                                    tmpEvent_anti[6] = 0; //X
                                    tmpEvent_anti[7] = y_pro_C1; //Y
                                    tmpEvent_anti[8] = z_pro_C1; //Z
                                    tmpSize_anti = 9;
                                }
                            }

                            
                            tetabra2 = asin(lamda / setup->tw_d2_anti);

                            if(setup->Curve_crystall){
                                double2 corrRes = Util_CUDA::horCorr(
                                    -y_pro_C1,
                                    setup->y_max_C2,
                                    z_pro_C1,
                                    setup->z_max_C2,
                                    false,
                                    setup->R_cur_crys_1,
                                    setup->R_cur_crys_2);

                                Costeta_CHC = corrRes.x;
                                Sinteta_CHC = corrRes.y;

                                n2x_anti = setup->n2x_anti_temp * Costeta_CHC + setup->n2y_anti_temp * Sinteta_CHC;
                                n2y_anti = setup->n2y_anti_temp * Costeta_CHC - setup->n2x_anti_temp * Sinteta_CHC;
                            }


                            double4 angleRes = Util_CUDA::getFullAngle2(r2x, r2y, r2z, n2x_anti, n2y_anti, setup->n2z_anti);
                            angle = angleRes.x;
                            r3x = angleRes.y;
                            r3y = angleRes.z;
                            r3z = angleRes.w;


                            Util_CUDA::ReflectionParameters parsR = {
                                setup->available_energies,
                                setup->min_angle_resp,
                                setup->max_angle_resp,
                                setup->mka_poli,
                                setup->Crystal_Responces
                            };

                            sec_crystal_Antiparallel_reach = Util_CUDA::getReflection(randState + idx + stride,
                                    angle,
                                    tetabra2,
                                    lamda,
                                    false,
                                    parsR,
                                    poliP);

                            if(sec_crystal_Antiparallel_reach){

                                if(setup->make_image_plates){
                                    if(setup->make_imageC2_After_refle){
                                        Util_CUDA::MakeParameters *pars = new Util_CUDA::MakeParameters(
                                                reduce[idx].max_plot_x,
                                                reduce[idx].max_plot_y,
                                                reduce[idx].hist_image_plate_source,
                                                reduce[idx].hist_image_plate_crystal1,
                                                reduce[idx].hist_image_plate_crystal2_para,
                                                reduce[idx].hist_image_plate_crystal2_anti,
                                                reduce[idx].hist_image_plate_detc_para,
                                                reduce[idx].hist_image_plate_detc_anti,
                                                reduce[idx].max_hist,
                                                reduce[idx].counts_sour,
                                                reduce[idx].counts_C1,
                                                reduce[idx].counts_C2_para,
                                                reduce[idx].counts_detc_para,
                                                reduce[idx].counts_C2_anti,
                                                reduce[idx].counts_detc_anti
                                        );
                                        
                                        Util_CUDA::Make(5, y_pro_C1, z_pro_C1, pars);
                                    }
                                }


                                rx_rot_sec = setup->cos_tetartabdete_anti * r3x + setup->sin_tetartabdete_anti * r3y;
                                ry_rot_sec = - setup->sin_tetartabdete_anti * r3x + setup->cos_tetartabdete_anti * r3y;


                                var_temp = - ry_rot_sec / rx_rot_sec;
                                tetadir_det = atan(var_temp);
                                tan_tetadir_det = var_temp;
                                cos_tetadir_det = cos(tetadir_det);


                                var_temp = rz / sqrt(pow(rx_rot, 2) + pow(ry_rot, 2));
                                fidir_det = atan(var_temp);
                                tan_fidir_det = var_temp;
                                cos_fidir_det = cos(fidir_det);
                                cos_fidirtilt2_anti = cos(fidir + setup->tilt_C2_anti_temp);

                                corr_dis_d_pa = y_pro_C1 * setup->cos_difteC2_det_para;
                                y_pro_C1_d_pa = y_pro_C1 * setup->sin_difteC2_det_para;


                                if(y_pro_C1_d_pa == 0){
                                    cos_tetap_det = 0;
                                    if(z_pro_C1 > 0)
                                        sin_tetap_det = 1;
                                    else
                                        sin_tetap_det = -1;
                                }else{
                                    var_temp = z_pro_C1 / y_pro_C1_d_pa;
                                    r_det = sqrt(pow(z_pro_C1, 2) + pow(y_pro_C1_d_pa, 2));

                                    if(y_pro_C1_d_pa > 0){
                                        sin_tetap_det = var_temp / sqrt(1 + pow(var_temp, 2));
                                        cos_tetap_det = 1 / sqrt(1 + pow(var_temp, 2));
                                    }else{
                                        sin_tetap_det = - var_temp / sqrt(1 + pow(var_temp, 2));
                                        cos_tetap_det = - 1 / sqrt(1 + pow(var_temp, 2));
                                    }
                                }


                                double2 yz = Util_CUDA::getYZ(r_det, sin_tetap_det, cos_tetap_det, tan_tetadir_det, tan_fidir_det, setup->dist_Cr2_det_Db - corr_dis_d_pa);

                                y_det = yz.x;
                                z_det = yz.y;

                                if(y_det < setup->ydetc_2_max && y_det > setup->ydetc_2_min && z_det < setup->zdetc_2_max && z_det > setup->zdetc_2_min){

                                    if (setup->make_image_plates) {
                                        Util_CUDA::MakeParameters *pars = new Util_CUDA::MakeParameters(
                                                reduce[idx].max_plot_x,
                                                reduce[idx].max_plot_y,
                                                reduce[idx].hist_image_plate_source,
                                                reduce[idx].hist_image_plate_crystal1,
                                                reduce[idx].hist_image_plate_crystal2_para,
                                                reduce[idx].hist_image_plate_crystal2_anti,
                                                reduce[idx].hist_image_plate_detc_para,
                                                reduce[idx].hist_image_plate_detc_anti,
                                                reduce[idx].max_hist,
                                                reduce[idx].counts_sour,
                                                reduce[idx].counts_C1,
                                                reduce[idx].counts_C2_para,
                                                reduce[idx].counts_detc_para,
                                                reduce[idx].counts_C2_anti,
                                                reduce[idx].counts_detc_anti
                                        );
                                        
                                        Util_CUDA::Make(6, y_det, z_det, pars);

                                        //Event point at detector in antiparallel
                                        tmpEvent_anti[9] = 0; //X
                                        tmpEvent_anti[10] = y_det; //Y
                                        tmpEvent_anti[11] = z_det; //Z
                                        tmpSize_anti = 12;

                                        if (bin->curr3DEventCount_anti <= setup->number_events) {
                                            reduce[idx].eventsToTrace_antiCUDA[bin->curr3DEventCount_anti] = tmpEvent_anti;
                                            atomicAdd(&(bin->curr3DEventCount_anti), 1);
                                        }
                                        else {
                                            for (int i = 0; i < bin->curr3DEventCount_anti; ++i)
                                                reduce[idx].eventsToTrace_antiCUDA[i] = reduce[idx].eventsToTrace_antiCUDA[i + 1];
                                            
                                            reduce[idx].eventsToTrace_antiCUDA[bin->curr3DEventCount_anti] = tmpEvent_anti;
                                        }
                                    }

                                    reduce[idx].toint_anti++;

                                    reduce[idx].energy_sum_anti_thread += Convert_Ag_minusone_eV / lamda;
                                }
                                else {
                                    if (setup->make_image_plates) {
                                        //If the event does not reach the detector then only add when we have less than maxEventNum
                                        if (bin->curr3DEventCount_anti <= setup->number_events && tmpSize_anti >= 6) {
                                            reduce[idx].eventsToTrace_antiCUDA[bin->curr3DEventCount_anti] = tmpEvent_anti;
                                            atomicAdd(&(bin->curr3DEventCount_anti), 1);
                                        }
                                    }
                                }

                            }
                            else {
                                if (setup->make_image_plates) {
                                    //If the event does not reach the detector then only add when we have less than maxEventNum
                                    if (bin->curr3DEventCount_anti <= setup->number_events && tmpSize_anti >= 6) {
                                        reduce[idx].eventsToTrace_antiCUDA[bin->curr3DEventCount_anti] = tmpEvent_anti;
                                        atomicAdd(&(bin->curr3DEventCount_anti), 1);
                                    }
                                }
                            }

                        }
                        else {
                            if (setup->make_image_plates) {
                                //If the event does not reach the detector then only add when we have less than maxEventNum
                                if (bin->curr3DEventCount_anti < setup->number_events && tmpSize_anti >= 6) {
                                    reduce[idx].eventsToTrace_antiCUDA[bin->curr3DEventCount_anti] = tmpEvent_anti;
                                    atomicAdd(&(bin->curr3DEventCount_anti), 1);
                                }
                            }
                        }
                    }
                }
            }
        }
        total_current_bins++;
    }
}